//sys
#include <cmath>
#include <stdio.h>
#include <cassert>
#include <iostream>
#include <hip/hip_runtime.h>
//my
#include "detect.h"

#define NV_CUDA_CHECK(status)                                                                      \
    {                                                                                              \
        if (status != 0)                                                                           \
        {                                                                                          \
            std::cout << "Cuda failure: " << hipGetErrorString(status) << " in file " << __FILE__ \
                      << " at line " << __LINE__ << std::endl;                                     \
            abort();                                                                               \
        }                                                                                          \
    }

namespace nvinfer1
{
	Detect::Detect(const void* data, size_t length)
	{

	}

	Detect::Detect(const uint32_t &n_anchor_, const uint32_t &n_classes_,
		const uint32_t &n_grid_h_, const uint32_t &n_grid_w_):
		_n_anchor(n_anchor_),
		_n_classes(n_classes_),
		_n_grid_h(n_grid_h_),
		_n_grid_w(n_grid_w_)
	{
		_n_output_size = (5 + _n_classes)*_n_anchor*_n_grid_h*_n_grid_w;
	}
	Detect::~Detect()
	{}

	int Detect::enqueue(int batchSize, const void* const* inputs, void** outputs, void* workspace,
		hipStream_t stream)
	{
		return 0;
	}

	size_t Detect::getSerializationSize() const
	{
		return sizeof(_n_anchor) + sizeof(_n_classes) + sizeof(_n_grid_h) + sizeof(_n_grid_w) + sizeof(_n_output_size);
	}

	void Detect::serialize(void *buffer) const
	{

	}

	void Detect::configurePlugin(const PluginTensorDesc* in, int nbInput, const PluginTensorDesc* out, int nbOutput) override
	{

	}
	IPluginV2IOExt* Detect::clone() const
	{
		Detect *p = new Detect(_n_anchor,_n_classes,_n_grid_h,_n_grid_w);
		p->setPluginNamespace(_s_plugin_namespace.c_str());
		return p;
	}
}//end namespace nvinfer1
