//sys
#include <cmath>
#include <stdio.h>
#include <cassert>
#include <iostream>
#include <hip/hip_runtime.h>
//my
#include "detect.h"

#define NV_CUDA_CHECK(status)                                                                      \
    {                                                                                              \
        if (status != 0)                                                                           \
        {                                                                                          \
            std::cout << "Cuda failure: " << hipGetErrorString(status) << " in file " << __FILE__ \
                      << " at line " << __LINE__ << std::endl;                                     \
            abort();                                                                               \
        }                                                                                          \
    }

namespace nvinfer1
{
	

}//end namespace nvinfer1
